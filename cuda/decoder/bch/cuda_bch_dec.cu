#include "hip/hip_runtime.h"
/* Cuda programming implementation of the bch decoder using Berkleykamp algorithm */
/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
// System includes
#include <stdio.h>
#include <assert.h>                           
#include <math.h>

/*
// CUDA runtime 
#include <hip/hip_runtime.h> 
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>    
*/

#define GPU_IMP
// Project related includes
#include <bch_cuda_defines.h>
#include <gf_defines.h>
#include <gf_bch_defines.h>
#include <gf_func.cu>

/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/

GFN_DEF void cuda_gf_init();
GFN_DEF void cuda_bch_syndrome(DTYPEP pg_data, UINTP syndrome);
GFN_DEF void cuda_bch_keyeq(UINTP syndrome, DTYPEP keyeq);
GFN_DEF void cuda_bch_csearch(DTYPEP keyeq,DTYPEP pg_data, DTYPEP pg_corr_data);

// Function to initialize the memory (DW) 
void memory_init (DTYPEP x,int N) {
  int i;
  for(i=0;i<N;i++) {
	 x[i] = i;
  }
}  


// Main call for the routine
int main() {
  int pg_size    = (F_BLOCK_SIZE/8)*F_NBLOCKS;
  int pg_size_dw = pg_size/SZ_OF_DTYPE;
  int pg_syn_sz  = F_NBLOCKS*2*T*4;
  hipError_t err = hipSuccess;

  /* Allocate memory for each block on the host end */
  DTYPEP h_pg_data       = (DTYPEP) malloc(pg_size);
  DTYPEP h_pg_corr_data  = (DTYPEP) malloc(pg_size);
  
  /* Alocate memory for the block on the GPU */
  DTYPEP d_pg_data;      CUDA_CHK_ERR(hipMalloc(&d_pg_data,pg_size));
  DTYPEP d_pg_corr_data; CUDA_CHK_ERR(hipMalloc(&d_pg_corr_data, pg_size));
  UINTP  d_pg_syndrome;  CUDA_CHK_ERR(hipMalloc(&d_pg_syndrome,pg_syn_sz));
  DTYPEP d_pg_keyeq;     CUDA_CHK_ERR(hipMalloc(&d_pg_keyeq,(T+1)*SZ_OF_DTYPE*F_NBLOCKS));
   
  /* Call a host initialization */
  memory_init (h_pg_data,pg_size_dw);
  memory_init (h_pg_corr_data,pg_size_dw);


  //++++++++++++++++++++++++++++++++++++++++++++++++++++++

  /* Copy the data from the host memory to the GPU */
  err = hipMemcpy (d_pg_data, h_pg_data, pg_size, hipMemcpyHostToDevice);
  CUDA_CHK_ERR(err);

  /* Cuda Kernel calls and associated variables */
  dim3 cuda_grid;
  dim3 cuda_block;

  cuda_grid  = dim3(1);cuda_block = dim3(1);
  cuda_gf_init CUDA_VEC ();
  #ifndef NOERR
  err = hipGetLastError();CUDA_CHK_ERR(err);
  #endif

  // The block and grid size cannot be more than 1024

  cuda_grid.x  = F_NBLOCKS;
  cuda_grid.y  = pg_size_dw/F_NBLOCKS;
  cuda_grid.z  = 1;
  cuda_block.x = (SZ_OF_DTYPE*8);
  cuda_block.y = 2*T; 
  cuda_block.z = 1;
  cuda_bch_syndrome CUDA_VEC (d_pg_data,d_pg_syndrome);
  #ifndef NOERR
  err = hipGetLastError();CUDA_CHK_ERR(err);
  #endif

  cuda_grid.x  = 1;cuda_grid.y  = 1;cuda_grid.z  = 1;
  cuda_block.x = F_NBLOCKS;
  cuda_block.y = 1;
  cuda_block.z = 1;
  cuda_bch_keyeq CUDA_VEC (d_pg_syndrome,d_pg_keyeq);
  #ifndef NOERR
  err = hipGetLastError();CUDA_CHK_ERR(err);
  #endif

  cuda_grid.x  = pg_size_dw/F_NBLOCKS;
  cuda_grid.y  = F_NBLOCKS;
  cuda_grid.z  = 1;
  cuda_block.x = (SZ_OF_DTYPE*8);
  cuda_block.y = 1;
  cuda_block.z = 1;
  cuda_bch_csearch CUDA_VEC (d_pg_keyeq,d_pg_data,d_pg_corr_data);
  #ifndef NOERR
  err = hipGetLastError();CUDA_CHK_ERR(err);
  #endif

  /* Once the computation is done, move the corrected data back to the host */
  err = hipMemcpy (h_pg_corr_data, d_pg_syndrome, pg_syn_sz, hipMemcpyDeviceToHost);
  #ifndef NOERR
  CUDA_CHK_ERR(err);
  #endif

  //++++++++++++++++++++++++++++++++++++++++++++++++++++++
  DTYPEP h_dbg = (DTYPEP) malloc ((1<<M)*4);
  //  err = hipMemcpyFromSymbol(h_dbg,HIP_SYMBOL(gb_gf_ext),((1<<M)*4));
  //  CUDA_CHK_ERR(err);

  // Final print 
/*
  for(i=0;i<pg_size_dw;i++){
	 printf("GF element %03d is %04x \n",i,h_pg_corr_data[i]);
  }
*/
  /* Free up the cuda memory */
  hipFree(d_pg_data);hipFree(d_pg_syndrome);hipFree(d_pg_corr_data);
  free(h_pg_data);free(h_pg_corr_data);
  
}

/* Subroutine to initialize the galois field element */
GFN_DEF void cuda_gf_init(){
  DTYPE i,elem;

  gb_gf_ext[0] = elem = 1;
  gb_gf_log_table[1] = gb_gf_log_table[0] = 0;

  for (i=1;i<(1<<M)-1;i++) {
	  elem = elem << 1;
    if (elem >= (1<<M)) {
		  elem = (elem ^ CS_PRIM_POLY[M]) & CS_GF_WND;
    }
	 gb_gf_ext[i] = elem;
    gb_gf_log_table[elem] = i;
  }
}



/* syndrome generator */
GFN_DEF void cuda_bch_syndrome (DTYPEP pg_data, UINTP syndrome){

  DTYPE dw_data_pos,dw_pos,block_pos,synd_i;
  DTYPE pow_i,bit_pos,bl_dw_pos,synd_pos,synd_calc_pos;
  

  // The position of the 32 bit is the thread id   
  bl_dw_pos = threadIdx.x;
  bit_pos   = threadIdx.y;
  synd_pos  = blockIdx.y;
  block_pos = blockIdx.x;

  dw_pos   = (blockDim.x * blockIdx.x) + threadIdx.x;
  synd_i   = synd_pos;

  dw_data_pos = pg_data[dw_pos] & (1<<bit_pos);

  synd_calc_pos = synd_i + (gridDim.y * block_pos);

  syndrome[synd_calc_pos] = 0;
  __syncthreads();

  pow_i = ((synd_i * (bl_dw_pos * SZ_OF_DTYPE*8))+bit_pos) % ((1<<M)-1); 

  if(dw_data_pos != 0) { atomicXor(&syndrome[synd_calc_pos],gb_gf_log_table[pow_i]);}
}

/* Key equation solver */
GFN_DEF void cuda_bch_keyeq (UINTP syndrome, DTYPEP keyeq) {
  DTYPE block_pos = threadIdx.x*(2*T*SZ_OF_UINT);

  DTYPE beta[T+1][T+1];
  DTYPE lr[T+1];
  DTYPE dp,dr,dp_cons,bsel;
  DTYPE sigma[T+1][T+1];
  DTYPE beta_mul[T+1];
 
  DTYPE s0 = syndrome[block_pos];

  /* First initialize the array */
  int i,r;
  for (i=0;i<=T;i++) {
  	 if(s0 != 0) {
		dp = s0;
		beta[1][i] = (i==2) ? 1 : 0;
		lr[1] =1;
	 } else {
		dp = 1;
		beta[1][i] = (i==3) ? 1 : 0;
		lr[1] =0;
	 }
	 sigma[0][i] = (i==0) ? 1 : (i==1) ? s0 : 0;
  }

  // The iteration is T times.
  for(r=1;r<T;r++) {
	 dr = 0;
	 for(i=0;i<T;i++) {
		dr = dr ^ gfn_gf_mul(sigma[r-1][i],syndrome[block_pos+(2*r-i)]);
	 }
	 dp_cons = dr;
    for(i=0;i<=T;i++){
		beta_mul[i] = gfn_gf_mul(beta[r][i],dp_cons);
	 }
	 for(i=0;i<=T;i++){
		sigma[r][i] = beta_mul[i] ^ gfn_gf_mul(sigma[r-1][i],dp);
	 }
	 bsel = (dr != 0 && r >= lr[r]) ? 1 : 0 ;
    for(i=0;i<=T;i++){
		beta[r+1][i+2] = bsel ? sigma[r-1][i] : beta[r][i];
	 }
	 beta[r+1][0] = beta[r+1][1] = 0;

    lr[r+1] = bsel ? lr[r]+1 : lr[r];
    dp = bsel ? dr : dp;
  }

  // Now assign the result back
  for(i=0;i<=T;i++) {
	 keyeq[block_pos+i] = sigma[T-1][i];
  }
}

/* Chein search algorithm to correct the errors */
GFN_DEF void cuda_bch_csearch (DTYPEP keyeq,DTYPEP pg_data,DTYPEP pg_corr_data) {
  DTYPE bit_pos   = threadIdx.x;
   
  DTYPE bl_pos    = blockIdx.x;
  DTYPE dw_pos    = blockDim.x * blockIdx.y + bl_pos;
  DTYPE key_pos   = blockIdx.y*(T+1);

  DTYPE sum,err_det;;
  DTYPE alpha_pos;
  DTYPE alpha_val;
  int i;

  // Find if the poistion has a solvable error 

  for (i=0;i<=T;i++) {
    alpha_pos = (bit_pos+(bl_pos*SZ_OF_DTYPE*8));
    alpha_pos = (alpha_pos*i)%CS_GF_WND;
	  alpha_val = gb_gf_log_table[alpha_pos];
	  sum = sum ^ gfn_gf_mul(keyeq[key_pos+i],alpha_val);
  }

  err_det = (sum != 0) ? 0 : 1;
  pg_data[dw_pos] = pg_data[dw_pos] ^ (err_det << bit_pos);

  /*
  pg_corr_data[dw_pos] = pg_corr_data[dw_pos] ^ (err_det << bit_pos);
  __syncthreads();  
  pg_corr_data[dw_pos] = pg_corr_data[dw_pos] ^ pg_data[dw_pos];
  */
}
