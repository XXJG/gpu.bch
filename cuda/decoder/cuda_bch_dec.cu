#include "hip/hip_runtime.h"
/* Cuda programming implementation of the bch decoder using Berkleykamp algorithm */
/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/

// System includes                                                                                                                                                                                                                                                               
#include <stdio.h>                                                                                                                                                                                                                                                               
#include <assert.h>                           
#include <math.h>                                                                                                                                                                                                                                   

// CUDA runtime                                                                                                                                                                                                                                                                  
#include <hip/hip_runtime.h>                                                                                                                                                                                                                                                        

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>    

// Include locally defined header file 
#include <bch_cuda_defines.h>
#include <gf_defines.h>
#include <gf_func.cu>

/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/


/* syndrome generator */
__global__ void cuda_bch_syndrome(UINTP d_pg_data, UINTP d_corr_data){
  UINT data_pos;
  UINT dw_data;
   
  data_pos = threadIdx.x;
  dw_data = d_pg_data[data_pos];
}


/* Subroutine to initialize the galois field element */
__global__ void cuda_gf_init(){
  UINT i,elem;

  gb_gf_ext[0] = elem = 1;
  gb_gf_log_table[1] = gb_gf_log_table[0] = 0;

  for (i=1;i<(1<<M)-1;i++) {
	 elem = elem << i;
    if (elem >= (1<<M)) {
      elem = (elem ^ cs_prim_poly[M]) & cs_gf_wind;
    }
	 gb_gf_ext[i] = elem;
    gb_gf_log_table[elem] = i;
  }
}


/* ++++++++++++++++++++++++++++++++++++++++++++++++++++++++++ 
  This is the main function to do the bch decoder algorithm 
  ++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
int main() {
  int pg_size_dw = ceil(BLOCK_SIZE/8)/sizeof(UINT)*NBLOCKS;

  /* Allocate memory for each block on the host end */
  UINTP h_pg_data       = (UINTP) malloc(pg_size_dw);
  UINTP h_pg_corr_data  = (UINTP) malloc(pg_size_dw);
  UINTP h_dw_bit_pos    = (UINTP) malloc(32);
  UINTP h_dw_sft_pos    = (UINTP) malloc(32);

  /* initialize the memory for the bit position and sft position */
  int i;
  for(i=0;i<32;i++){ h_dw_bit_pos[i] = int(1<<i);h_dw_sft_pos[i] = i;}

  /* Alocate memory for the block on the GPU */
  UINTP d_pg_data;      hipMalloc(&d_pg_data,pg_size_dw);
  UINTP d_pg_syndrome;  hipMalloc(&d_pg_syndrome,2*T*NBLOCKS);
  UINTP d_pg_corr_data; hipMalloc(&d_pg_corr_data, pg_size_dw);

  /* Copy the data from the host memory to the GPU */
  hipMemcpy(d_pg_data, h_pg_data, pg_size_dw, hipMemcpyHostToDevice);

  /* Initialize the table */
  cuda_gf_init<<<1,1>>>(); // This is done only once


  /* Invoke the kernel with a single thread */
  dim3 cuda_threads(1,2*T,pg_size_dw/NBLOCKS);
  int cuda_blocks  = NBLOCKS;
      
  cuda_bch_syndrome<<<cuda_blocks,cuda_threads>>>(d_pg_data,d_pg_syndrome);
  
  /* Once the computation is done, move the corrected data back to the host */
  hipMemcpy(h_pg_corr_data, d_pg_corr_data, pg_size_dw, hipMemcpyDeviceToHost);

  printf("storage for the unsigned int is %d",sizeof(UINTP));

}

