#include "hip/hip_runtime.h"
/* Cuda programming implementation of the bch decoder using Berkleykamp algorithm */
/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/

// System includes                                                                                                                                                                                                                                                               
#include <stdio.h>                                                                                                                                                                                                                                                               
#include <assert.h>                           
#include <math.h>                                                                                                                                                                                                                                   

// CUDA runtime                                                                                                                                                                                                                                                                  
#include <hip/hip_runtime.h>                                                                                                                                                                                                                                                        

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>    

// Include locally defined header file 
#include <bch_cuda_defines.h>
#include <gf_defines.h>
#include <gf_func.cu>

/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/


/* syndrome generator */
__global__ void cuda_bch_syndrome(UINTP d_pg_data, UINTP d_corr_data){
  UINT data_pos;
  UINT dw_data;
   
  data_pos = threadIdx.x;
  dw_data = d_pg_data[data_pos];
}

void gf_init();


/* ++++++++++++++++++++++++++++++++++++++++++++++++++++++++++ 
  This is the main function to do the bch decoder algorithm 
  ++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
int main() {
  int pg_size_dw = ceil(BLOCK_SIZE/8)/sizeof(UINT)*NBLOCKS;
  int i;

  /* Allocate memory for each block on the host end */
  UINTP h_pg_data       = (UINTP) malloc(pg_size_dw);
  UINTP h_pg_corr_data  = (UINTP) malloc(pg_size_dw);
  
  /* Alocate memory for the block on the GPU */
  UINTP d_pg_data;      hipMalloc(&d_pg_data,pg_size_dw);
  UINTP d_pg_syndrome;  hipMalloc(&d_pg_syndrome,2*T*NBLOCKS);
  UINTP d_pg_corr_data; hipMalloc(&d_pg_corr_data, pg_size_dw);

  /* Copy the data from the host memory to the GPU */
  hipMemcpy(d_pg_data, h_pg_data, pg_size_dw, hipMemcpyHostToDevice);

  /* Initialize the table */
  //  cuda_gf_init<<<1,1>>>(); // This is done only once
  gf_init(); // This is done only once

  /* Invoke the kernel with a single thread */
  dim3 cuda_threads(1,2*T,pg_size_dw/NBLOCKS);
  int cuda_blocks  = NBLOCKS;
      
  cuda_bch_syndrome<<<cuda_blocks,cuda_threads>>>(d_pg_data,d_pg_syndrome);
  
  /* Once the computation is done, move the corrected data back to the host */
  hipMemcpy(h_pg_corr_data, d_pg_corr_data, pg_size_dw, hipMemcpyDeviceToHost);


  /* Print values on the host side */
  //UINTP h_dbg = (UINTP) malloc(1<<M);
  UINTP h_dbg;

  //  hipMemcpy(h_dbg, gb_gf_ext, (1<<M) , hipMemcpyDeviceToHost); 
  h_dbg = gf_ext;
 
  for(i=0;i<(1<<M);i++){
	 printf("GF element %03d is %08x \n",i,h_dbg[i]);
  }

  /* Free up the cuda memory */
  hipFree(d_pg_data);hipFree(d_pg_syndrome);hipFree(d_pg_corr_data);
  
}


/* Subroutine to initialize the galois field element */
//__global__ void cuda_gf_init(){
void gf_init(){
  UINT i,elem;

  gf_ext[0] = elem = 1;
  gf_log_table[1] = gf_log_table[0] = 0;

  for (i=1;i<(1<<M)-1;i++) {
	 elem = elem << 1;
    if (elem >= (1<<M)) {
		//      elem = (elem ^ cs_prim_poly[M]) & cs_gf_wind;
      elem = (elem ^ 0x201d) & ((1<<M) -1);
    }
	 gf_ext[i] = elem;
    gf_log_table[elem] = i;
  }
}

