/* Cuda programming implementation of the bch decoder using Berkleykamp algorithm */
/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/

// System includes                                                                                                                                                                                                                                                               
#include <stdio.h>                                                                                                                                                                                                                                                               
#include <assert.h>                                                                                                                                                                                                                                                              

// CUDA runtime                                                                                                                                                                                                                                                                  
#include <hip/hip_runtime.h>                                                                                                                                                                                                                                                        

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>    

// Include local definitions of the file
#include <bch_def.h>

/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/


/* This is the main function to do the bch decoder algorithm */
int main() {
  int array_size = BLOCK_SIZE/sizeof(UINT);

  /* Allocate memory for each block on the host end */
  UINTP h_block_data = (UINTP) malloc(array_size);
  UINTP h_corr_data  = (UINTP) malloc(array_size);

  /* Alocate memory for the block on the GPU */
  UINTP d_block_data; hipMalloc(&d_block_data,array_size);
  UINTP d_corr_data ; hipMalloc(&d_corr_data, array_size);

  /* Copy the data from the host memory to the GPU */
  hipMemcpy(d_block_data, h_block_data, array_size, hipMemcpyHostToDevice);

  /* Invoke the kernel with a single thread */
  int num_of_threads = 1;
  int num_of_blocks  = 1;

  
  /* Once the computation is done, move the corrected data back to the host */
  hipMemcpy(h_corr_data, d_corr_data, array_size, hipMemcpyDeviceToHost);

  printf("storage for the unsigned int is %d",sizeof(UINTP));

}
