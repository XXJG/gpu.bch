#include "hip/hip_runtime.h"
/* Cuda programming implementation of the bch decoder using Berkleykamp algorithm */
/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/

// System includes                                                                                                                                                                                                                                                               
#include <stdio.h>                                                                                                                                                                                                                                                               
#include <assert.h>                           
#include <math.h>                                                                                                                                                                                                                                   

// CUDA runtime                                                                                                                                                                                                                                                                  
#include <hip/hip_runtime.h>                                                                                                                                                                                                                                                        

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>    

// Include locally defined header file 
#include <bch_cuda_defines.h>
#include <gf_defines.h>
#include <gf_func.cu>

/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/


/* syndrome generator */
__global__ void cuda_bch_syndrome(UINTP d_pg_data, UINTP d_corr_data){

  UINT dw_data;
  int data_pos;
   
  data_pos = threadIdx.x;
  dw_data = d_pg_data[data_pos];
  d_corr_data[data_pos] = d_pg_data[data_pos];
}

/* Define the function here, so could be used in the main program*/
__global__ void cuda_gf_init();


/* ++++++++++++++++++++++++++++++++++++++++++++++++++++++++++ 
  This is the main function to do the bch decoder algorithm 
  ++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
int main() {
  int pg_size = (BLOCK_SIZE/8/sizeof(UINT))*NBLOCKS;
  int pg_size_dw  = pg_size/4; 
  int i;

  /* Allocate memory for each block on the host end */
  UINTP h_pg_data       = (UINTP) malloc(pg_size);
  UINTP h_pg_corr_data  = (UINTP) malloc(pg_size);
  
  /* Alocate memory for the block on the GPU */
  UINTP d_pg_data;      hipMalloc(&d_pg_data,pg_size);
  UINTP d_pg_syndrome;  hipMalloc(&d_pg_syndrome,2*T*NBLOCKS);
  UINTP d_pg_corr_data; hipMalloc(&d_pg_corr_data, pg_size);

  for(i=0;i<pg_size_dw;i++) {
	 h_pg_data[i] = i;
	 h_pg_corr_data[i] = i;
  }

  /* Copy the data from the host memory to the GPU */
  hipMemcpy(d_pg_data, h_pg_data, pg_size, hipMemcpyHostToDevice);

  /* Initialize the table */
  //cuda_gf_init<<<1,1>>>(); // This is done only once
  //gf_init(); // This is done only once

  /* Invoke the kernel with a single thread */
  //  dim3 cuda_threads(2*T,pg_size_dw/NBLOCKS);
  int cuda_threads = 512;
  int cuda_blocks  = NBLOCKS;
      
  cuda_bch_syndrome<<<cuda_blocks,cuda_threads>>>(d_pg_data,d_pg_corr_data);

  /* Once the computation is done, move the corrected data back to the host */
  hipMemcpy(h_pg_corr_data, d_pg_corr_data, pg_size, hipMemcpyDeviceToHost);


  /* Print values on the host side */
  UINTP h_dbg = (UINTP) malloc(1<<M);

  //hipMemcpy(h_dbg, &cg_gf_ext, (1<<M) , hipMemcpyDeviceToHost); 
  // h_dbg = gf_ext;

 
  for(i=0;i<(1<<M);i++){
	 printf("GF element %03d is %04x \n",i,h_pg_data[i]);
	 printf("GF element %03d is %04x \n",i,h_pg_corr_data[i]);
  }

  /* Free up the cuda memory */
  hipFree(d_pg_data);hipFree(d_pg_syndrome);hipFree(d_pg_corr_data);
  
}


/* Subroutine to initialize the galois field element */
__global__ void cuda_gf_init(){
// void gf_init(){
  UINT i,elem;

  cg_gf_ext[0] = elem = 1;
  cg_gf_log_table[1] = cg_gf_log_table[0] = 0;

  for (i=1;i<(1<<M)-1;i++) {
	 elem = elem << 1;
    if (elem >= (1<<M)) {
		elem = (elem ^ ccs_prim_poly[M]) & ccs_gf_wind;
    }
	 cg_gf_ext[i] = elem;
    cg_gf_log_table[elem] = i;
  }
}

