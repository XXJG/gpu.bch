#include "hip/hip_runtime.h"
/* Cuda programming implementation of the bch decoder using Berkleykamp algorithm */
/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
// System includes
#include <stdio.h>                                                                                                                                                                                                                                                           #include <assert.h>                           
#include <math.h>                                                                                                                                                                                                                                   

// CUDA runtime                                                                                                                                                                                                                                                              #include <hip/hip_runtime.h>                                                                                                                                                                                                                                                    
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>    

#define GPU_IMP
// Project related includes
#include <bch_cuda_defines.h>
#include <gf_defines.h>
#include <gf_func.cu>

/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/

GFN_DEF void cuda_gf_init();
GFN_DEF void cuda_bch_syndrome(UINTP pg_data, UINTP corr_data);

// Function to initialize the memory (DW) 
void memory_init (UINTP x,int N) {
  int i;
  for(i=0;i<N;i++) {
	 x[i] = i;
  }
}  


// Main call for the routine
int main() {
  int pg_size    = (BLOCK_SIZE/8)*NBLOCKS;
  int pg_size_dw = ceil(pg_size/sizeof(UINT)); 
  int i;
  hipError_t err = hipSuccess;

  /* Allocate memory for each block on the host end */
  UINTP h_pg_data       = (UINTP) malloc(pg_size);
  UINTP h_pg_corr_data  = (UINTP) malloc(pg_size);
  
  /* Alocate memory for the block on the GPU */
  UINTP d_pg_data;      CUDA_CHK_ERR(hipMalloc(&d_pg_data,pg_size));
  UINTP d_pg_syndrome;  CUDA_CHK_ERR(hipMalloc(&d_pg_syndrome,2*T*NBLOCKS));
  UINTP d_pg_corr_data; CUDA_CHK_ERR(hipMalloc(&d_pg_corr_data, pg_size));

  /* Call a host initialization */
  memory_init (h_pg_data,pg_size_dw);
  memory_init (h_pg_corr_data,pg_size_dw);


  //++++++++++++++++++++++++++++++++++++++++++++++++++++++

  /* Copy the data from the host memory to the GPU */
  hipMemcpy (d_pg_data, h_pg_data, pg_size, hipMemcpyHostToDevice);

  /* Cuda Kernel calls and associated variables */
  dim3 cuda_grid;
  dim3 cuda_block;

  cuda_grid  = dim3(1);cuda_block = dim3(1);
  cuda_gf_init CUDA_VEC ();
  err = hipGetLastError();CUDA_CHK_ERR(err);

  // The block and grid size cannot be more than 1024

  cuda_grid.x  = 2*T;    
  cuda_grid.y  = NBLOCKS;   
  cuda_grid.z  = 1;
  cuda_block.x = pg_size_dw; 
  cuda_block.y = 1; 
  cuda_block.z = 1;
  cuda_bch_syndrome CUDA_VEC (d_pg_data,d_pg_corr_data);
  err = hipGetLastError();CUDA_CHK_ERR(err);

  /* Once the computation is done, move the corrected data back to the host */
  hipMemcpy (h_pg_corr_data, d_pg_corr_data, pg_size, hipMemcpyDeviceToHost);

  //++++++++++++++++++++++++++++++++++++++++++++++++++++++

  // Final print 
  for(i=0;i<pg_size_dw;i++){
	 printf("GF element %03d is %04x \n",i,h_pg_data[i]);
	 printf("GF element %03d is %04x \n",i,h_pg_corr_data[i]);
  }

  /* Free up the cuda memory */
  hipFree(d_pg_data);hipFree(d_pg_syndrome);hipFree(d_pg_corr_data);
  
}


/* Subroutine to initialize the galois field element */
GFN_DEF void cuda_gf_init(){
  UINT i,elem;

  gb_gf_ext[0] = elem = 1;
  gb_gf_log_table[1] = gb_gf_log_table[0] = 0;

  for (i=1;i<(1<<M)-1;i++) {
	 elem = elem << 1;
    if (elem >= (1<<M)) {
		elem = (elem ^ CS_PRIM_POLY[M]) & CS_GF_WND;
    }
	 gb_gf_ext[i] = elem;
    gb_gf_log_table[elem] = i;
  }
}

/* syndrome generator */
GFN_DEF void cuda_bch_syndrome(UINTP pg_data, UINTP corr_data){

  UINT dw_data;
  int data_pos;
   
  data_pos = threadIdx.x;
  dw_data  = pg_data[data_pos];

  corr_data[data_pos] = pg_data[data_pos];
}
