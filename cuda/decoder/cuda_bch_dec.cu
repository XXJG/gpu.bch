/* Cuda programming implementation of the bch decoder using Berkleykamp algorithm */
/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
// System includes
#include <stdio.h>                                                                                                                                                                                                                                                           #include <assert.h>                           
#include <math.h>                                                                                                                                                                                                                                   

// CUDA runtime                                                                                                                                                                                                                                                              #include <hip/hip_runtime.h>                                                                                                                                                                                                                                                    
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>    

#define GPU_IMP
// Project related includes
#include <bch_cuda_defines.h>
#include <gf_defines.h>
#include <gf_func.cu>

/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/

GFN_DEF void cuda_gf_init();
GFN_DEF void cuda_bch_syndrome(UINTP pg_data, UINTP corr_data);

// Function to initialize the memory (DW) 
void memory_init (UINTP x,int N) {
  int i;
  for(i=0;i<N;i++) {
	 x[i] = i;
  }
}  


// Main call for the routine
int main() {
  int pg_size    = (BLOCK_SIZE/8)*NBLOCKS;
  int pg_size_dw = pg_size/SZ_OF_UINT;
  int i;
  hipError_t err = hipSuccess;

  /* Allocate memory for each block on the host end */
  UINTP h_pg_data       = (UINTP) malloc(pg_size);
  UINTP h_pg_corr_data  = (UINTP) malloc(pg_size);
  
  /* Alocate memory for the block on the GPU */
  UINTP d_pg_data;      CUDA_CHK_ERR(hipMalloc(&d_pg_data,pg_size));
  UINTP d_pg_syndrome;  CUDA_CHK_ERR(hipMalloc(&d_pg_syndrome,2*T*NBLOCKS));
  UINTP d_pg_corr_data; CUDA_CHK_ERR(hipMalloc(&d_pg_corr_data, pg_size));
   
  /* Call a host initialization */
  memory_init (h_pg_data,pg_size_dw);
  memory_init (h_pg_corr_data,pg_size_dw);


  //++++++++++++++++++++++++++++++++++++++++++++++++++++++

  /* Copy the data from the host memory to the GPU */
  err = hipMemcpy (d_pg_data, h_pg_data, pg_size, hipMemcpyHostToDevice);
  CUDA_CHK_ERR(err);

  /* Cuda Kernel calls and associated variables */
  dim3 cuda_grid;
  dim3 cuda_block;

  cuda_grid  = dim3(1);cuda_block = dim3(1);
  cuda_gf_init CUDA_VEC ();
  err = hipGetLastError();CUDA_CHK_ERR(err);

  // The block and grid size cannot be more than 1024

  cuda_grid.x  = 2*T;    
  cuda_grid.y  = NBLOCKS;
  cuda_grid.z  = 1;
  cuda_block.x = pg_size_dw/NBLOCKS; 
  cuda_block.y = 1; 
  cuda_block.z = 1;
  cuda_bch_syndrome CUDA_VEC (d_pg_data,d_pg_corr_data);
  err = hipGetLastError();CUDA_CHK_ERR(err);

  /* Once the computation is done, move the corrected data back to the host */
  err = hipMemcpy (h_pg_corr_data, d_pg_corr_data, pg_size, hipMemcpyDeviceToHost);
  CUDA_CHK_ERR(err);

  //++++++++++++++++++++++++++++++++++++++++++++++++++++++
  UINTP h_dbg = (UINTP) malloc ((1<<M)*4);
  //  err = hipMemcpyFromSymbol(h_dbg,HIP_SYMBOL(gb_gf_ext),((1<<M)*4));
  //  CUDA_CHK_ERR(err);

  // Final print 
  for(i=0;i<pg_size_dw;i++){
  // for(i=0;i<(1<<M);i++){
	 //	 printf("GF element %03d is %04x \n",i,h_dbg[i]);
	 printf("GF element %03d is %04x \n",i,h_pg_corr_data[i]);
  }

  /* Free up the cuda memory */
  hipFree(d_pg_data);hipFree(d_pg_syndrome);hipFree(d_pg_corr_data);
  
}


/* Subroutine to initialize the galois field element */
GFN_DEF void cuda_gf_init(){
  UINT i,elem;

  gb_gf_ext[0] = elem = 1;
  gb_gf_log_table[1] = gb_gf_log_table[0] = 0;

  for (i=1;i<(1<<M)-1;i++) {
	  elem = elem << 1;
    if (elem >= (1<<M)) {
		  elem = (elem ^ CS_PRIM_POLY[M]) & CS_GF_WND;
    }
	 gb_gf_ext[i] = elem;
    gb_gf_log_table[elem] = i;
  }
}



/* syndrome generator */
GFN_DEF void cuda_bch_syndrome(UINTP pg_data, UINTP synd_mult){

  UINT dw_data,dw_pos,synd_i,alpha_pow_i;;
  UINT pow_i;

  // The position of the 32 bit is the thread id   
  dw_pos   = (blockDim.x * blockIdx.y) + threadIdx.x;
  synd_i   = blockIdx.x;
  dw_data  = pg_data[dw_pos];
  synd_mult[dw_pos] = 0;
  int i;

  // For loop for adding up the size of bits
  for(i=0;i<SZ_OF_UINT;i++) {
	 pow_i = ((synd_i * dw_pos)+i) % ((1<<M)-1); 
	 if(dw_data & (1<<(i+0))) { synd_mult[dw_pos] ^= gb_gf_log_table[pow_i];}
  }
}
