#include "hip/hip_runtime.h"
/* Cuda programming implementation of the reed solomon decoder */
/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
// System includes
#include <stdio.h>   
#include <assert.h>                           
#include <math.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>  

#define GPU_IMP
// Project related includes
#include <bch_cuda_defines.h>
#include <gf_defines.h>
#include <gf_rs_defines.h>
#include <gf_func.cu>

/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
/* Cuda Kernel calls and associated variables */
dim3 cuda_grid;
dim3 cuda_block;

/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/

GFN_DEF void cuda_gf_init();
GFN_DEF void cuda_rs_syndrome(DTYPEP pg_data, UINTP syndrome);
GFN_DEF void cuda_rs_keyeq(UINTP syndrome, DTYPEP keyeq);
GFN_DEF void cuda_rs_csearch(DTYPEP keyeq,DTYPEP pg_data, DTYPEP pg_corr_data);

// Function to initialize the memory (DW) 
void memory_init (DTYPEP x,int N) {
  int i;
  for(i=0;i<N;i++) {
	 x[i] = i;
  }
}  

// Main call for the routine
int main() {
  int bl_sz     = 256;
  int bl_sz_dw  = bl_sz/SZ_OF_DTYPE;
  // int bl_syn_sz = 2*T*(SZ_OF_DTYPE); 
  int bl_syn_sz = 2*T;
  hipError_t err = hipSuccess;

  /* Allocate memory for each block on the host end */
  DTYPEP h_pg_data       = (DTYPEP) malloc(bl_sz);
  DTYPEP h_pg_corr_data  = (DTYPEP) malloc(bl_sz);
  
  /* Alocate memory for the block on the GPU */
  DTYPEP d_pg_data;      CUDA_CHK_ERR(hipMalloc(&d_pg_data,bl_sz));
  DTYPEP d_pg_corr_data; CUDA_CHK_ERR(hipMalloc(&d_pg_corr_data, bl_sz));
  UINTP  d_pg_syndrome;  CUDA_CHK_ERR(hipMalloc(&d_pg_syndrome,bl_syn_sz));
  DTYPEP d_pg_keyeq;     CUDA_CHK_ERR(hipMalloc(&d_pg_keyeq,(T+1)*SZ_OF_DTYPE));
   
  /* Call a host initialization */
  memory_init (h_pg_data,bl_sz_dw);
  memory_init (h_pg_corr_data,bl_sz_dw);


  //++++++++++++++++++++++++++++++++++++++++++++++++++++++
  /* Copy the data from the host memory to the GPU */
  err = hipMemcpy (d_pg_data, h_pg_data, bl_sz, hipMemcpyHostToDevice);
  CUDA_CHK_ERR(err);


  cuda_grid  = dim3(1);cuda_block = dim3(1);
  cuda_gf_init CUDA_VEC ();
  err = hipGetLastError();CUDA_CHK_ERR(err);

  // The block and grid size cannot be more than 1024
  cuda_grid.x  = 2*T; 
  cuda_grid.y  = 1; 
  cuda_grid.z  = 1;
  cuda_block.x = RS_N;
  cuda_block.y = 1; 
  cuda_block.z = 1;
  cuda_rs_syndrome CUDA_VEC (d_pg_data,d_pg_syndrome);
  err = hipGetLastError();CUDA_CHK_ERR(err);

  // Call berlekamp massey algorithm
  cuda_grid.x  = 1;cuda_grid.y  = 1;cuda_grid.z  = 1;
  cuda_block.x = 1;
  cuda_block.y = 1;
  cuda_block.z = 1;
  cuda_rs_keyeq CUDA_VEC (d_pg_syndrome,d_pg_keyeq);
  err = hipGetLastError();CUDA_CHK_ERR(err);

  cuda_grid.x  = F_NBLOCKS;
  cuda_grid.y  = F_NBLOCKS;
  cuda_grid.z  = 1;
  cuda_block.x = SZ_OF_DTYPE;
  cuda_block.y = 1;
  cuda_block.z = 1;
  cuda_rs_csearch CUDA_VEC (d_pg_keyeq,d_pg_data,d_pg_corr_data);
  err = hipGetLastError();CUDA_CHK_ERR(err);

  /* Once the computation is done, move the corrected data back to the host */
  err = hipMemcpy (h_pg_corr_data, d_pg_corr_data, bl_sz, hipMemcpyDeviceToHost);
  CUDA_CHK_ERR(err);

  //++++++++++++++++++++++++++++++++++++++++++++++++++++++
  DTYPEP h_dbg = (DTYPEP) malloc ((1<<M)*4);
  //  err = hipMemcpyFromSymbol(h_dbg,HIP_SYMBOL(gb_gf_ext),((1<<M)*4));
  //  CUDA_CHK_ERR(err);

  // Final print 
/*
  for(i=0;i<gl_sz;i++){
	 printf("GF element %03d is %04x \n",i,h_pg_corr_data[i]);
  }
*/
  /* Free up the cuda memory */
  hipFree(d_pg_data);hipFree(d_pg_syndrome);hipFree(d_pg_corr_data);
  free(h_pg_data);free(h_pg_corr_data);
  
}


/* Subroutine to initialize the galois field element */
GFN_DEF void cuda_gf_init(){
  DTYPE i,elem;

  gb_gf_ext[0] = elem = 1;
  gb_gf_log_table[1] = gb_gf_log_table[0] = 0;

  for (i=1;i<(1<<M)-1;i++) {
	  elem = elem << 1;
    if (elem >= (1<<M)) {
		  elem = (elem ^ CS_PRIM_POLY[M]) & CS_GF_WND;
    }
	 gb_gf_ext[i] = elem;
    gb_gf_log_table[elem] = i;
  }
}



/* syndrome generator */
GFN_DEF void cuda_rs_syndrome (DTYPEP pg_data, UINTP syndrome){

  __shared__ int l_syndrome[RS_N];

  // The position of the 32 bit is the thread id   
  DTYPE bl_dw_pos  = threadIdx.x;
  DTYPE byte_pos   = threadIdx.y;
  DTYPE synd_i     = blockIdx.x;
  DTYPE block_no   = blockIdx.y;

  DTYPE dw_pos        = bl_dw_pos; 
  DTYPE dw_data       = pg_data[dw_pos];
  DTYPE byte          = dw_data>>(byte_pos*8) & 0xff; 
  DTYPE synd_calc_pos = synd_i + 0; 

  DTYPE m       = gb_gf_log_table[byte];
  DTYPE pow_i  = ((synd_i * bl_dw_pos)+m) % ((1<<M)-1); 

  int log_table = gb_gf_ext[(pow_i)];
  // atomicXor(&l_syndrome[synd_calc_pos],log_table);

  l_syndrome[bl_dw_pos] = log_table; // Assign the multiplied value to the syndrome
  __syncthreads(); // This will make sure the array is synchronized 
  

  int index = RS_N; 
  index /= 2; 

  while (index != 0) {
    // Galois field addition is XOR
    if (bl_dw_pos < index) { l_syndrome[bl_dw_pos] ^= l_syndrome[bl_dw_pos+index];}

    __syncthreads(); // This will make sure the values are synchronized 
    index = index/2;
  }
  

  syndrome[synd_i] = l_syndrome[synd_i];
}

/* Key equation solver */
GFN_DEF void cuda_rs_keyeq (UINTP syndrome, DTYPEP keyeq) {
  DTYPE block_pos = threadIdx.x*(2*T*SZ_OF_UINT);

  DTYPE beta[T+1][T+1];
  DTYPE lr[T+1];
  DTYPE dp,dr,dp_cons,bsel;
  DTYPE sigma[T+1][T+1];
  DTYPE beta_mul[T+1];
 
  DTYPE s0 = syndrome[block_pos];

  /* First initialize the array */
  int i,r;
  for (i=0;i<=T;i++) {
  	 if(s0 != 0) {
		dp = s0;
		beta[1][i] = (i==2) ? 1 : 0;
		lr[1] =1;
	 } else {
		dp = 1;
		beta[1][i] = (i==3) ? 1 : 0;
		lr[1] =0;
	 }
	 sigma[0][i] = (i==0) ? 1 : (i==1) ? s0 : 0;
  }

  // The iteration is T times.
  for(r=1;r<T;r++) {
	 dr = 0;
	 for(i=0;i<T;i++) {
		dr = dr ^ gf_mul(sigma[r-1][i],syndrome[block_pos+(2*r-i)]);
	 }
	 dp_cons = dr;
    for(i=0;i<=T;i++){
		beta_mul[i] = gf_mul(beta[r][i],dp_cons);
	 }
	 for(i=0;i<=T;i++){
		sigma[r][i] = beta_mul[i] ^ gf_mul(sigma[r-1][i],dp);
	 }
	 bsel = (dr != 0 && r >= lr[r]) ? 1 : 0 ;
    for(i=0;i<=T;i++){
		beta[r+1][i+2] = bsel ? sigma[r-1][i] : beta[r][i];
	 }
	 beta[r+1][0] = beta[r+1][1] = 0;

    lr[r+1] = bsel ? lr[r]+1 : lr[r];
    dp = bsel ? dr : dp;
  }

  // Now assign the result back
  for(i=0;i<=T;i++) {
	 keyeq[block_pos+i] = sigma[T-1][i];
  }
}

/* Chein search algorithm to correct the errors */
GFN_DEF void cuda_rs_csearch (DTYPEP keyeq,DTYPEP pg_data,DTYPEP pg_corr_data) {
  DTYPE bit_pos   = threadIdx.x;
   
  DTYPE bl_pos    = blockIdx.x;
  DTYPE dw_pos    = blockDim.x * blockIdx.y + bl_pos;
  DTYPE key_pos   = blockIdx.y*(T+1);

  DTYPE err_mask = 0;
  DTYPE sum,err_det;;
  DTYPE alpha_pos;
  DTYPE alpha_val;
  DTYPE mult=0;
  int i;

  // Find if the poistion has a solvable error 

  for (i=0;i<=T;i++) {
    alpha_pos = (bit_pos+(bl_pos*SZ_OF_DTYPE));
    alpha_pos = (alpha_pos*i)%CS_GF_WND;
	 alpha_val = gb_gf_log_table[alpha_pos];
	 sum = sum ^ gf_mul(keyeq[key_pos+i],alpha_val);
  }

  err_det = (sum != 0) ? 0 : 1;
  pg_data[dw_pos] = pg_data[dw_pos] ^ (err_det << bit_pos);

  /*
  pg_corr_data[dw_pos] = pg_corr_data[dw_pos] ^ (err_det << bit_pos);
  __syncthreads();  
  pg_corr_data[dw_pos] = pg_corr_data[dw_pos] ^ pg_data[dw_pos];
  */
}
