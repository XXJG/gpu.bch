#include "hip/hip_runtime.h"
/* Cuda programming implementation of the reed solomon decoder */
/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
// System includes
#include <stdio.h>   
#include <assert.h>                           
#include <math.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>  

#define GPU_IMP
// Project related includes
#include <bch_cuda_defines.h>
#include <gf_defines.h>
#include <gf_rs_defines.h>
#include <gf_func.cu>

/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
/* Cuda Kernel calls and associated variables */
dim3 cuda_grid;
dim3 cuda_block;

/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/

GFN_DEF void cuda_gf_init();
GFN_DEF void cuda_rs_fft(DTYPEP t_data, UINTP f_data);
GFN_DEF void cuda_rs_keyeq(UINTP syndrome, DTYPEP keyeq);
GFN_DEF void cuda_rs_csearch(DTYPEP keyeq,DTYPEP pg_data, DTYPEP pg_corr_data);

// Function to initialize the memory (DW) 
void memory_init (DTYPEP x,int N) {
  int i;
  for(i=0;i<N;i++) {
	 x[i] = i;
  }
}  

// Main call for the routine
int main() {
  hipError_t err = hipSuccess;

  /* Allocate memory for each block on the host end */
  DTYPEP h_pg_data       = (DTYPEP) malloc(F_CPG_SIZE_BYTES);
  DTYPEP h_pg_corr_data  = (DTYPEP) malloc(F_CPG_SIZE_BYTES);
  
  /* Alocate memory for the block on the GPU */
  DTYPEP d_pg_data;      CUDA_CHK_ERR(hipMalloc(&d_pg_data,F_CPG_SIZE_BYTES));
  DTYPEP d_pg_corr_data; CUDA_CHK_ERR(hipMalloc(&d_pg_corr_data,F_CPG_SIZE_BYTES));
  UINTP  d_pg_syndrome;  CUDA_CHK_ERR(hipMalloc(&d_pg_syndrome,2*T*F_NO_OF_SC*4));
  UINTP  d_pg_rs_rx_fft; CUDA_CHK_ERR(hipMalloc(&d_pg_rs_rx_fft,RS_N*F_NO_OF_SC*4));
  DTYPEP d_pg_keyeq;     CUDA_CHK_ERR(hipMalloc(&d_pg_keyeq,(T+1)*F_NO_OF_SC*4));
   
  /* Call a host initialization */
  memory_init (h_pg_data,F_CPG_SIZE_BYTES/4);
  memory_init (h_pg_corr_data,F_CPG_SIZE_BYTES/4);


  //++++++++++++++++++++++++++++++++++++++++++++++++++++++
  /* Copy the data from the host memory to the GPU */
  err = hipMemcpy (d_pg_data, h_pg_data, F_CPG_SIZE_BYTES, hipMemcpyHostToDevice);
  CUDA_CHK_ERR(err);


  cuda_grid  = dim3(1);cuda_block = dim3(1);
  cuda_gf_init CUDA_VEC ();
  err = hipGetLastError();CUDA_CHK_ERR(err);

  printf ("Galois field is %3d error is %3d block size is %6d \n",M,T,RS_N);

#ifndef ERR_SEEN
  // The block and grid size cannot be more than 1024
  cuda_grid.x  = 2*T; 
  cuda_grid.y  = F_NO_OF_SC;
  cuda_grid.z  = 1;
  cuda_block.x = RS_N;
  cuda_block.y = 1; 
  cuda_block.z = 1;
  cuda_rs_fft CUDA_VEC (d_pg_data,d_pg_rs_rx_fft);
  err = hipGetLastError();CUDA_CHK_ERR(err);

#else
  // Run FFT on the complete spectrum
  cuda_grid.x  = RS_N;
  cuda_grid.y  = F_NO_OF_SC; 
  cuda_grid.z  = 1;
  cuda_block.x = RS_N;
  cuda_block.y = 1; 
  cuda_block.z = 1;
  cuda_rs_fft CUDA_VEC (d_pg_data,d_pg_rs_rx_fft);
  err = hipGetLastError();CUDA_CHK_ERR(err);

  // Call berlekamp massey algorithm
  cuda_grid.x  = 1;cuda_grid.y  = 1;cuda_grid.z  = 1;
  cuda_block.x = 1;cuda_block.y = 1;cuda_block.z = 1;
  cuda_rs_keyeq CUDA_VEC (d_pg_rs_rx_fft,d_pg_keyeq);
  err = hipGetLastError();CUDA_CHK_ERR(err);

  cuda_grid.x  = F_NO_OF_SC;
  cuda_grid.y  = F_NO_OF_SC;
  cuda_grid.z  = 1;
  cuda_block.x = 4;
  cuda_block.y = 1;
  cuda_block.z = 1;
  cuda_rs_csearch CUDA_VEC (d_pg_keyeq,d_pg_data,d_pg_corr_data);
  err = hipGetLastError();CUDA_CHK_ERR(err);
#endif

  /* Once the computation is done, move the corrected data back to the host */
  err = hipMemcpy (h_pg_corr_data, d_pg_corr_data, F_CPG_SIZE_BYTES, hipMemcpyDeviceToHost);
  CUDA_CHK_ERR(err);

  //++++++++++++++++++++++++++++++++++++++++++++++++++++++
  DTYPEP h_dbg = (DTYPEP) malloc ((1<<M)*4);
  //  err = hipMemcpyFromSymbol(h_dbg,HIP_SYMBOL(gb_gf_ext),((1<<M)*4));
  //  CUDA_CHK_ERR(err);

  // Final print 
/*
  for(i=0;i<gl_sz;i++){
	 printf("GF element %03d is %04x \n",i,h_pg_corr_data[i]);
  }

*/

   
  /* Free up the cuda memory */
  hipFree(d_pg_data);hipFree(d_pg_syndrome);hipFree(d_pg_corr_data);
  free(h_pg_data);free(h_pg_corr_data);
  
}


/* Subroutine to initialize the galois field element */
GFN_DEF void cuda_gf_init(){
  DTYPE i,elem;

  gb_gf_ext[0] = elem = 1;
  gb_gf_log_table[1] = gb_gf_log_table[0] = 0;

  for (i=1;i<(1<<M)-1;i++) {
	  elem = elem << 1;
    if (elem >= (1<<M)) {
		  elem = (elem ^ CS_PRIM_POLY[M]) & CS_GF_WND;
    }
	 gb_gf_ext[i] = elem;
    gb_gf_log_table[elem] = i;
  }
}



/* FFT in the given finite field */
GFN_DEF void cuda_rs_fft (DTYPEP t_data, UINTP f_data){

  __shared__ int l_f_data[RS_N];

  // The position of the 32 bit is the thread id   
  DTYPE bl_dw_pos  = threadIdx.x;
  DTYPE byte_pos   = threadIdx.y;
  DTYPE synd_i     = blockIdx.x;
  DTYPE block_no   = blockIdx.y;

  DTYPE elem          = bl_dw_pos; 
  DTYPE vec_elem      = t_data[block_no*RS_N+elem];
  DTYPE synd_calc_pos = synd_i + 0; 

  DTYPE m       = gb_gf_log_table[vec_elem];
  DTYPE pow_i  = ((synd_i * elem)+m) % ((1<<M)-1); 

  int log_table = gb_gf_ext[(pow_i)];

  l_f_data[elem] = log_table; // Assign the multiplied value to the syndrome
  __syncthreads(); // This will make sure the array is synchronized 
  

  int index = RS_N; 
  index /= 2; 

  while (index != 0) {
    // Galois field addition is XOR
    if (elem < index) { l_f_data[elem] ^= l_f_data[elem+index];}
    index = index/2;
    __syncthreads(); // This will make sure the values are synchronized 
  }
  
  f_data[block_no*RS_N+synd_i] = l_f_data[synd_i];
}

/* Key equation solver */
GFN_DEF void cuda_rs_keyeq (UINTP syndrome, DTYPEP keyeq) {
  DTYPE block_pos = threadIdx.x*(2*T*SZ_OF_UINT);

  DTYPE beta[T+1][T+1];
  DTYPE lr[T+1];
  DTYPE dp,dr,dp_cons,bsel;
  DTYPE sigma[T+1][T+1];
  DTYPE beta_mul[T+1];
 
  DTYPE s0 = syndrome[block_pos];

  /* First initialize the array */
  int i,r;
  for (i=0;i<=T;i++) {
  	 if(s0 != 0) {
		dp = s0;
		beta[1][i] = (i==2) ? 1 : 0;
		lr[1] =1;
	 } else {
		dp = 1;
		beta[1][i] = (i==3) ? 1 : 0;
		lr[1] =0;
	 }
	 sigma[0][i] = (i==0) ? 1 : (i==1) ? s0 : 0;
  }

  // The iteration is T times.
  for(r=1;r<T;r++) {
	 dr = 0;
	 for(i=0;i<T;i++) {
		dr = dr ^ gf_mul(sigma[r-1][i],syndrome[block_pos+(2*r-i)]);
	 }
	 dp_cons = dr;
    for(i=0;i<=T;i++){
		beta_mul[i] = gf_mul(beta[r][i],dp_cons);
	 }
	 for(i=0;i<=T;i++){
		sigma[r][i] = beta_mul[i] ^ gf_mul(sigma[r-1][i],dp);
	 }
	 bsel = (dr != 0 && r >= lr[r]) ? 1 : 0 ;
    for(i=0;i<=T;i++){
		beta[r+1][i+2] = bsel ? sigma[r-1][i] : beta[r][i];
	 }
	 beta[r+1][0] = beta[r+1][1] = 0;

    lr[r+1] = bsel ? lr[r]+1 : lr[r];
    dp = bsel ? dr : dp;
  }

  // Now assign the result back
  for(i=0;i<=T;i++) {
	 keyeq[block_pos+i] = sigma[T-1][i];
  }
}

/* Chein search algorithm to correct the errors */
GFN_DEF void cuda_rs_csearch (DTYPEP keyeq,DTYPEP pg_data,DTYPEP pg_corr_data) {
  DTYPE bit_pos   = threadIdx.x;
   
  DTYPE bl_pos    = blockIdx.x;
  DTYPE dw_pos    = blockDim.x * blockIdx.y + bl_pos;
  DTYPE key_pos   = blockIdx.y*(T+1);

  DTYPE err_mask = 0;
  DTYPE sum,err_det;;
  DTYPE alpha_pos;
  DTYPE alpha_val;
  DTYPE mult=0;
  int i;

  // Find if the poistion has a solvable error 

  for (i=0;i<=T;i++) {
    alpha_pos = (bit_pos+(bl_pos*SZ_OF_DTYPE));
    alpha_pos = (alpha_pos*i)%CS_GF_WND;
	 alpha_val = gb_gf_log_table[alpha_pos];
	 sum = sum ^ gf_mul(keyeq[key_pos+i],alpha_val);
  }

  err_det = (sum != 0) ? 0 : 1;
  pg_data[dw_pos] = pg_data[dw_pos] ^ (err_det << bit_pos);

  /*
  pg_corr_data[dw_pos] = pg_corr_data[dw_pos] ^ (err_det << bit_pos);
  __syncthreads();  
  pg_corr_data[dw_pos] = pg_corr_data[dw_pos] ^ pg_data[dw_pos];
  */
}
