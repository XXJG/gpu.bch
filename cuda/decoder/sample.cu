#include "hip/hip_runtime.h"
//++++++++++++++++++++++++++++++++++++++++++++++++++++++
/* Cuda implementation of the BCH algorithm with 
   Berleykamp inversionless algorith  
*/
//++++++++++++++++++++++++++++++++++++++++++++++++++++++

// System includes                                                                                                                                                                                                                                                               
#include <stdio.h>                                                                                                                                                                                                                                                               
#include <assert.h>                           
#include <math.h>                                                                                                                                                                                                                                   

// CUDA runtime                                                                                                                                                                                                                                                                  
#include <hip/hip_runtime.h>                                                                                                                                                                                                                                                        

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>    

#define UINTP float*
#define UINT  float
// Include locally defined header file 
//#include <bch_cuda_defines.h>
#include <gf_defines.h>
//#include <gf_func.cu>

#define CUDA_CHECK_ERR(err) \
  if(err != hipSuccess) { \
	 fprintf(stderr,"Got error on Cuda: (%s)\n",hipGetErrorString(err)); \
    exit(EXIT_FAILURE); \
}

__global__ void cuda_gf_init(UINTP d_pg_data, UINTP d_pg_corr_data) {
  int pos = threadIdx.x;
  
  //  d_pg_corr_data[pos] = d_pg_data[pos];
  d_pg_corr_data[0] = 10; 
  d_pg_corr_data[1] = 10; 
}

int main() {
  size_t pg_size = 8*sizeof(UINT);
  int pg_size_dw = pg_size/sizeof(UINT);
  int i;
  hipError_t err = hipSuccess;

  UINTP h_pg_data = (UINTP) malloc(pg_size);
  UINTP h_pg_corr_data = (UINTP) malloc(pg_size);
  UINTP test  = (UINTP) malloc(pg_size);

  for(i=0;i<pg_size_dw;i++) {
	 h_pg_data[i] = i*2;
   h_pg_corr_data[i] = test[i] = i*2;
  }

  UINTP d_pg_data; 
  err = hipMalloc((void **)&d_pg_data,pg_size); CUDA_CHECK_ERR(err);

  UINTP d_pg_corr_data; 
  err = hipMalloc((void **)&d_pg_corr_data,pg_size); CUDA_CHECK_ERR(err);

  err = hipMemcpy(d_pg_data, h_pg_data, pg_size, hipMemcpyHostToDevice);
  CUDA_CHECK_ERR(err);

  //  cuda_gf_init<<<1,pg_size_dw>>>(d_pg_data,d_pg_corr_data);
  cuda_gf_init<<<1,1>>>(d_pg_data,d_pg_corr_data);
  err = hipGetLastError();
  CUDA_CHECK_ERR(err);

  err = hipMemcpy(h_pg_corr_data, d_pg_corr_data, pg_size, hipMemcpyDeviceToHost);
  CUDA_CHECK_ERR(err);
  

  for(i=0;i<pg_size_dw;i++){
	 printf("Host going in element %03d is ---> %f \n",i,h_pg_data[i]);
	 printf("Host coming out element %03d is %f \n",i,h_pg_corr_data[i]);
	 printf("Host coming out test %03d is %f \n\n",i,test[i]);
  }

  return 0;
  
}
