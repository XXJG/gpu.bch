#include "hip/hip_runtime.h"
//++++++++++++++++++++++++++++++++++++++++++++++++++++++
/* Cuda implementation of the BCH algorithm with 
   Berleykamp inversionless algorith  
*/
//++++++++++++++++++++++++++++++++++++++++++++++++++++++

// System includes                                                                                                                                                                                                                                                               
#include <stdio.h>                                                                                                                                                                                                                                                               
#include <assert.h>                           
#include <math.h>                                                                                                                                                                                                                                   

// CUDA runtime                                                                                                                                                                                                                                                                  
#include <hip/hip_runtime.h>                                                                                                                                                                                                                                                        

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>    

#define UINTP float*
#define UINT  float
// Include locally defined header file 
//#include <bch_cuda_defines.h>
#include <gf_defines.h>
//#include <gf_func.cu>


__global__ void cuda_gf_init(UINTP d_pg_data, UINTP d_pg_corr_data) {
  int pos = threadIdx.x;
  
  //  d_pg_corr_data[pos] = d_pg_data[pos];
  d_pg_corr_data[0] = 10; 
  d_pg_corr_data[1] = 10; 


}

int main() {
  size_t pg_size = 8*sizeof(UINT);
  int pg_size_dw = pg_size/sizeof(UINT);
  int i;
  hipError_t err = hipSuccess;

  UINTP h_pg_data = (UINTP) malloc(pg_size);
  UINTP h_pg_corr_data = (UINTP) malloc(pg_size);
  UINTP test  = (UINTP) malloc(pg_size);

  for(i=0;i<pg_size_dw;i++) {
	 h_pg_data[i] = i*2;
   h_pg_corr_data[i] = test[i] = i*2;
  }

  UINTP d_pg_data; err = hipMalloc((void **)&d_pg_data,pg_size);
  if(err != hipSuccess) {
	 fprintf(stderr,"Failed to allocate memory for d_pg_data (err code %s)\n",hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  UINTP d_pg_corr_data; err = hipMalloc((void **)&d_pg_corr_data,pg_size);

  err = hipMemcpy(d_pg_data, h_pg_data, pg_size, hipMemcpyHostToDevice);
  if(err != hipSuccess) {
	 fprintf(stderr,"Failed to copy device to host (err code %s)\n",hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  //  cuda_gf_init<<<1,pg_size_dw>>>(d_pg_data,d_pg_corr_data);
  cuda_gf_init<<<1,1>>>(d_pg_data,d_pg_corr_data);
  err = hipGetLastError();

  err = hipMemcpy(h_pg_corr_data, d_pg_corr_data, pg_size, hipMemcpyDeviceToHost);
  if(err != hipSuccess) {
	 fprintf(stderr,"Failed to copy device to host (err code %s)\n",hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  

  for(i=0;i<pg_size_dw;i++){
	 printf("Host going in element %03d is ---> %f \n",i,h_pg_data[i]);
	 printf("Host coming out element %03d is %f \n",i,h_pg_corr_data[i]);
	 printf("Host coming out test %03d is %f \n\n",i,test[i]);
  }


  
}
