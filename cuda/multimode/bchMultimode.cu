/* Cuda programming implementation of the bch decoder using Berkleykamp algorithm */
/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
// System includes
#include <stdio.h>
#include <assert.h>                           
#include <math.h>

// CUDA runtime 
#include <hip/hip_runtime.h> 
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>    

#define GPU_IMP

// Project related includes
#include <bch_cuda_defines.h>
#include <gf_defines.h>
#include <gf_func.cu>

// Definition specific to this program
#define MAX_MEM_VALUE 10
#ifdef DBG
#define DBG_FLG 1 
#else
#define DBG_FLG 0 
#endif

/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/

GFN_DEF void cuda_gf_init();
GFN_DEF void cuda_bch_encoder(DTYPEP block_data);
GFN_DEF void cuda_bch_syndrome(DTYPEP pg_data, UINTP syndrome);
GFN_DEF void cuda_bch_keyeq(UINTP syndrome, DTYPEP keyeq);
GFN_DEF void cuda_bch_csearch(DTYPEP keyeq,DTYPEP pg_data, DTYPEP pg_corr_data);

// Function to initialize the memory (DW) 
void memory_init (DTYPEP x,int N) {
  int i;
  for(i=0;i<N;i++) {
	 x[i] = i % MAX_MEM_VALUE; 
  }
}  

// Main call for the routine
int main() {
  char DBG_MSG[100]="BCH_MULTIMODE";

  UINT c_block_size = BLOCK_SIZE; 

  if(DBG_FLG){
    printf("%s: Ext_field is            : %-4d \n",DBG_MSG,M);
    printf("%s: No of Error correction  : %-4d \n",DBG_MSG,T);
    printf("%s: Primitive polynomial is : %-#x \n",DBG_MSG,PRIMITIVE_POLY);
    printf("%s: Primitive XOR is        : %-#x \n",DBG_MSG,P_XOR);
    printf("%s: block_size in bytes     : %-#d \n",DBG_MSG,c_block_size);
  }

  hipError_t err = hipSuccess;

  /* Cuda Kernel calls and associated variables */
  dim3 cuda_thread; // This cannot exceed more than 1024
  dim3 cuda_block;  // This can be more than 1024

  // Initialize the table for the 
  cuda_block  = dim3(1);cuda_thread = dim3(1);
  cuda_gf_init CUDA_VEC ();
  err = hipGetLastError();CUDA_CHK_ERR(err);
}

/* Subroutine to initialize the galois field element */
GFN_DEF void cuda_gf_init(){
  DTYPE i,elem;

  gb_gf_ext[0] = elem = 1;
  gb_gf_log_table[1] = gb_gf_log_table[0] = 0;

  for (i=1;i<(1<<M)-1;i++) {
	  elem = elem << 1;
    if (elem >= (1<<M)) {
		  elem = (elem ^ DCS_PRIM_POLY[M]) & DCS_GF_WND;
    }
	 gb_gf_ext[i] = elem;
    gb_gf_log_table[elem] = i;
  }
}



