#include "hip/hip_runtime.h"
/* Functions that can be used for the galois field arithmetic */

/* Function to add two elements  */
__host__ __device__ UINT gfn_gf_add (UINT x, UINT y ) { return(x ^ y ); }

/* Function to generate the next galois field element */
__host__ __device__ UINT gfn_nxt_gf_element(UINT x) {
  x = x << 1; 
  if (x & (1 << M)) {
    x = (x & GF_MASK) ^ P_XOR; 
  }
  return x;
}

// Function to generate the multiplication with log table
__host__ __device__ UINT gfn_gf_log_mul(UINT x, UINT y) {
  x = x & GF_MASK;
  y = y & GF_MASK;
  if(x==0 || y == 0) { return 0; }

  #ifdef GPU_IMP
  UINT alpha_x,alpha_y,sum;
  alpha_x = gb_gf_log_table[x];
  alpha_y = gb_gf_log_table[y];
  sum = (alpha_x + alpha_y) % GF_MASK; 
  return gb_gf_ext[sum];
  #else
  return 0; 
  #endif
}

// Function to generate the multiplication with the standard basis
__host__ __device__ UINT gfn_gf_mul(UINT x, UINT y) {
  if(x==0 || y == 0) { return 0; }
  UINT r,t,i;

  for(i=0;i<M;i++){
    
    if(i==0 ) {
      r = (y & 1) ? x : 0;
      t = x;
    } else {
      t = gfn_nxt_gf_element(t);
      y = y >> 1;
      r = (y & 1) ? t ^ r : r; 
    } 
  }
  return r;
}

__global__  void dfn_gf_mul_ker(UINT x, UINT y, UINTP r) {
 int bit_id = threadIdx.x;
 atomicXor(r,x);  
}
