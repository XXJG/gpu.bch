#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>

#define N 10

// kernel routine
__global__ void sub_kernel(int* d_vec, int n) {
  int tid = threadIdx.x;
  if(threadIdx.x < n) {
    int i = d_vec[tid];
    d_vec[tid] = i > 5 ? -i : i;
  }
}

__global__ void kernel(int* d_vec, int n) {
  if(threadIdx.x < n) {
    sub_kernel<<<1,5>>>(d_vec,n);
  }
  hipDeviceSynchronize();
}


// Print routine 
void myprint (int *vec, int n) {
  int i;
  for(i=0;i<n;i++) {
    printf("vec [%d] = %d\n",i,*(vec+i));
  }
}

int main (int argc, char** argv) {
  int vec_in[N] = {7, 1, 2, 3, 6, 8, 10, 5, 4, 9};
  int vec_out[N];
  int* d_vec;

  myprint(&vec_in[0],N);

  // Allocate memory 
  hipMalloc(&d_vec, N*sizeof(int));
  hipMemcpy(d_vec,vec_in,N*sizeof(int),hipMemcpyHostToDevice);
  kernel<<<512,8>>>(d_vec,N);
  hipMemcpy(vec_out,d_vec,N*sizeof(int),hipMemcpyDeviceToHost); 

  myprint(&vec_out[0],N);

  hipFree(d_vec);
  return 0;
}
