#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>

// Add specific modules
#include <bch_cuda_defines.h>
#include <gf_defines.h>
#include <gf_bch_defines.h>
#include <gf_func.cu>

// Routine specific
#define N 11

// Sub kernel
__global__ void sub_kernel(UINTP d_vec_in, UINTP d_vec_out, UINT n) {
  int this_id = threadIdx.x;
  if(this_id < n) {
    d_vec_out[this_id] = gfn_gf_mul(d_vec_in[this_id],d_vec_in[this_id]);;
  }
  hipDeviceSynchronize();
}

// kernel routine
__global__ void kernel(UINTP d_vec_in, UINTP d_vec_out, UINT n) {
  sub_kernel<<<1,32>>>(d_vec_in,d_vec_out,n);
}

// Print routine 
void myprint (UINTP vec, int n) {
  int i;
  for(i=0;i<n;i++) {
    printf("vec [%d] = %x\n",i,*(vec+i));
  }
}

// Main program for the routine
int main (int argc, char** argv) {
  int i; 

  UINT  vec_out[N];
  // UINT  vec_in[N];
  UINT vec_in[N] = { 0b0001, 0b0010, 0b0100, 0b1000, 0b0011, 0b0110, 0b1100, 0b1011, 0b0101, 0b1010, 0b0111};

  for (i=0;i<N;i++) {
    vec_out[i] = 0;  
  }

  UINTP d_vec_in;
  UINTP d_vec_out;

  printf("Afer intialization");
  myprint(&vec_in[0],N);

  // Allocate memory 
  CUDA_CHK_ERR(hipMalloc(&d_vec_in, N*sizeof(UINT)));
  CUDA_CHK_ERR(hipMalloc(&d_vec_out,N*sizeof(UINT)));

  CUDA_CHK_ERR(hipMemcpy(d_vec_in,vec_in,N*sizeof(UINT),hipMemcpyHostToDevice));
  kernel<<<32,512>>>(d_vec_in,d_vec_out,N);
  CUDA_CHK_ERR(hipMemcpy(vec_out,d_vec_out,N*sizeof(UINT),hipMemcpyDeviceToHost)); 

  myprint(&vec_in[0],N);
  myprint(&vec_out[0],N);

  hipFree(d_vec_in);
  hipFree(d_vec_out);
  return 0;
}
